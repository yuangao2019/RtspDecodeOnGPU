#include "hip/hip_runtime.h"
#include "CudaFunction.h"

__global__ void
yuv420p2bgr32_kernel(hipDeviceptr_t yuv_buffer_in, cv::cuda::PtrStepSz<uchar3> rgb_buffer_out, unsigned int pitch,
                     int height, int width, int32_t real_height, int32_t real_width) {
	unsigned char* yuv_buffer = (unsigned char*) yuv_buffer_in;

	int           channels = 3;
	int           index_Y;
	int           index_U;
	int           index_V;
	unsigned char Y;
	unsigned char U;
	unsigned char V;

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	index_Y = y * pitch + x;
	Y       = yuv_buffer[index_Y];

	if (x % 2 == 0) {
		index_U = (height + y / 2) * pitch + x;
		index_V = (height + y / 2) * pitch + x + 1;
		U       = yuv_buffer[index_U];
		V       = yuv_buffer[index_V];
	} else if (x % 2 == 1) {
		index_V = (height + y / 2) * pitch + x;
		index_U = (height + y / 2) * pitch + x - 1;
		U       = yuv_buffer[index_U];
		V       = yuv_buffer[index_V];
	}

	// YCbCr420
	int R = Y + 1.402 * (V - 128);
	int G = Y - 0.34413 * (U - 128) - 0.71414 * (V - 128);
	int B = Y + 1.772 * (U - 128);


	// 确保取值范围在 0 - 255 中
	R = (R < 0) ? 0 : R;
	G = (G < 0) ? 0 : G;
	B = (B < 0) ? 0 : B;
	R = (R > 255) ? 255 : R;
	G = (G > 255) ? 255 : G;
	B = (B > 255) ? 255 : B;

	if (x < real_width && y < real_height) {
		rgb_buffer_out(y, x) = {(unsigned char) B, (unsigned char) G, (unsigned char) R};
	}
}

void yuv420p2bgr32(hipDeviceptr_t yuv_buffer_in, cv::cuda::GpuMat& rgb_buffer_out, unsigned int pitch,
                   int height, int width, int32_t real_height, int32_t real_width){
//	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 gridSize((real_width + 16 - 1) / 16, (real_height + 16 - 1) / 16);
	dim3 blockSize(16, 16);
	yuv420p2bgr32_kernel<<< gridSize, blockSize >>>(yuv_buffer_in, rgb_buffer_out, pitch, height, width, real_height, real_width);
	hipDeviceSynchronize();
}